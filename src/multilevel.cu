#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h> 
#include <time.h> 
#include <random>
#include <vector> 
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "timer.h"
#include "cuda_error_check.h"
#include "alloc.h"
#include "reduce_block_1d.h"
#include "complex.h"

#include "parameters.h"
#include "index.h"
#include "staple.h"
#include "random.h"


#include "tune.h"

#include "array.h"
#include "multilevel.h"
#include "lattice_functions.h"

using namespace std;

namespace U1{


#include "multilevel_common.cuh"

__global__ void kernel_l2_multilevel_1(complexd *poly, complexd *l2, int Rmax){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;    
	if(id >= SpatialVolume()) return;		
	int x[4];
	indexNOSD(id, x);
	
	int nlayers = Grid(TDir())/2;
	for(int r = 0; r < Rmax; ++r){	
		for(int dir = 0; dir < TDir(); dir++){		
			int layer = 0;
			for(int t = 0; t < Grid(TDir()); t+=2){
				complexd pl0 = 1.;
				complexd pl1 = 1.;
				for(x[TDir()] = t; x[TDir()] < t+2; ++x[TDir()]){
					pl0 *= (poly[indexId(x)]);
					int xold = x[dir];
					x[dir] = (x[dir] + r) % Grid(dir);
					pl1 *= conj(poly[indexId(x)]);
									
					x[dir] = xold;
				}			
				complexd pl= pl0 * pl1;
				//int pos = id + SpatialVolume() * layer + nlayers * SpatialVolume() * (r-1) + nlayers * SpatialVolume() * Rmax * dir;			
				int pos = id + SpatialVolume() * r + SpatialVolume() * Rmax * dir + SpatialVolume() * Rmax * (Dirs()-1) * layer;
				l2[pos] = pl + l2[pos];
				layer++;
			}
		}
	}
}



class L2ML: Tunable{
private:
	Array<complexd> *poly;
	Array<complexd> *l2;
	size_t sl2;
	int Rmax;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_l2_multilevel_1<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(poly->getPtr(), l2->getPtr(), Rmax);
}
public:	
   L2ML(Array<complexd> *poly, Array<complexd> *l2, size_t sl2, int Rmax) : poly(poly), l2(l2), sl2(sl2), Rmax(Rmax) {
	size = SpatialVolume();
	timesec = 0.0;  
}
   ~L2ML(){ };
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_get_time(){	return timesec;}
   void stat(){	cout << "L2ML:  " <<  get_get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {
  	l2->Backup();	
  }
  void postTune() {  
	l2->Restore();
 }

};





__global__ void kernel_l2avg_l4_multilevel(complexd *dev_l2, complexd *dev_l4, int Rmax, double l2norm){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    size_t size = SpatialVolume() * Rmax * (Dirs()-1);
    if(id >= size) return;			
	
	int nl2 = Grid(TDir())/2;
	int l4 = 0;
	for(int l2 = 0; l2 < nl2; l2+=2){
		complexd pl = 1.;
		for(int layer = l2; layer < l2+2; ++layer){
			int newid = id + size * layer;
			pl *= dev_l2[newid] * l2norm;
		}
		int pos = id + size * l4;
		dev_l4[pos] = pl + dev_l4[pos];
		l4++;	
	}
}


class L2AvgL4ML: Tunable{
private:
	Array<complexd> *l4;
	Array<complexd> *l2;
	double l2norm;
	size_t sl4;
	int Rmax;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_l2avg_l4_multilevel<<<tp.grid, tp.block, 0, stream>>>(l2->getPtr(), l4->getPtr(), Rmax, l2norm);
}
public:	
   L2AvgL4ML(Array<complexd> *l2, Array<complexd> *l4, size_t sl4, int Rmax, double l2norm) : l2(l2), l4(l4), sl4(sl4), Rmax(Rmax), l2norm(l2norm) {
	size = SpatialVolume() * Rmax * (Dirs()-1);
	timesec = 0.0;  
}
   ~L2AvgL4ML(){ };
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_get_time(){	return timesec;}
   void stat(){	cout << "L2AvgL4ML:  " <<  get_get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {
  	l4->Backup();
  }
  void postTune() {  
  	l4->Restore();
 }

};


template<bool savePPspace>
__global__ void kernel_l4avg_Final_multilevel(complexd *dev_l4, complexd *res, complexd *ppSpace, int Rmax, double norm){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;    				
	
	int nl4 = Grid(TDir())/4;	
	for(int r = 0; r < Rmax; ++r)	{
		complexd pp = 0.;
		if( id < SpatialVolume() ){
			for(int dir = 0; dir < TDir(); dir++){
				complexd pl = 1.;
				for(int l4 = 0; l4 < nl4; ++l4){
					int newid = id + SpatialVolume() * r + SpatialVolume() * Rmax * dir + SpatialVolume() * Rmax * (Dirs()-1) * l4;
					pl *= dev_l4[newid] * norm;
				}
				pp += pl;
				if(savePPspace) ppSpace[id + SpatialVolume() * dir + SpatialVolume() * (Dirs()-1) * r] = pl;
			}
		}
		reduce_block_1d<complexd>(res + r, pp);
		__syncthreads();
	}
}

template<bool savePPspace>
class L4AvgPP: Tunable{
private:
	Array<complexd> *l4;
	Array<complexd> *dev_poly;
	Array<complexd> *poly;
	int Rmax;
	double norm;
	double l4norm;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return sizeof(complexd); }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	dev_poly->Clear();
	if(savePPspace) kernel_l4avg_Final_multilevel<savePPspace><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(l4->getPtr(), dev_poly->getPtr(), ppSpace->getPtr(), Rmax, l4norm);
	else kernel_l4avg_Final_multilevel<savePPspace><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(l4->getPtr(), dev_poly->getPtr(), 0, Rmax, l4norm);
	
}
public:
	Array<complexd> *ppSpace;
	Array<complexd>* getField(){ return ppSpace; }
	
   L4AvgPP(Array<complexd> *l4, int Rmax, double l4norm) : l4(l4), Rmax(Rmax), l4norm(l4norm) {
	size = SpatialVolume();
	dev_poly = new Array<complexd>(Device, Rmax);
	if(savePPspace) ppSpace = new Array<complexd>(Device, SpatialVolume() * Rmax * (Dirs()-1));
	poly = new Array<complexd>(Host, Rmax);
	norm = 1. / double(SpatialVolume()*(Dirs()-1));
	timesec = 0.0;  
}
   ~L4AvgPP(){ delete dev_poly; };
   Array<complexd>* Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
	poly->Copy(dev_poly);
	for(int i = 0; i < Rmax; ++i) poly->getPtr()[i] *= norm;
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   Array<complexd>* Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_get_time(){	return timesec;}
   void stat(){	cout << "L4AvgPP:  " <<  get_get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};




Array<complexd>* MultiLevel(Array<double> *lat, CudaRNG *rng_state, int n4, int k4, int n2, int k2, int metrop, int ovrn, int Rmax, bool PrintResultsAtEveryN4){
	Timer a0; a0.start();

	cout << "==============================================" << endl;
	cout << "Rmax: " << Rmax << endl;
	cout << "----------------------------------------------" << endl;
	cout << "Level 0:" << endl;
	cout << "\tNº time links per slice: " << 2 << endl;
	cout << "\tNº iterations: " << n2 << endl;
	cout << "\tNº updates: " << k2 << endl;
	cout << "\tNº metropolis updates: " << metrop << endl;
	cout << "\tNº overrelaxation updates: " << ovrn << endl;
	cout << "----------------------------------------------" << endl;
	cout << "Level 1:" << endl;
	cout << "\tNº time links per slice: " << 4 << endl;
	cout << "\tNº iterations: " << n4 << endl;
	cout << "\tNº updates: " << k4 << endl;
	cout << "\tNº metropolis updates: " << metrop << endl;
	cout << "\tNº overrelaxation updates: " << ovrn << endl;
	cout << "==============================================" << endl;
	
	if( Grid(TDir())%4 != 0 ) {
		cout << "Error: Cannot Apply MultiLevel Algorithm...\n Nt is not multiple of 4...\n Exiting..." << endl;
		exit(1);
	}
	Array<double>* dev_lat = new Array<double>(Device);
	dev_lat->Copy(lat);

	int nl2 = Grid(TDir())/2;
	int sl2 = nl2*(Dirs()-1)*Rmax*SpatialVolume();
	Array<complexd> *l2 = new Array<complexd>(Device, sl2);
	int nl4 = Grid(TDir())/4;
	size_t sl4 = nl4*(Dirs()-1)*Rmax*SpatialVolume();
	Array<complexd> *l4 = new Array<complexd>(Device, sl4);
	
	// metropolis and overrelaxation algorithm
	Metropolis_ML<4> mtp4(dev_lat, rng_state, metrop);
	OverRelaxation_ML<4> ovr4(dev_lat, ovrn);
	
	Metropolis_ML<2> mtp2(dev_lat, rng_state, metrop);
	OverRelaxation_ML<2> ovr2(dev_lat, ovrn);
	
	const bool multihit = true;
	Polyakov_Volume<multihit> mhitVol(dev_lat);
	Array<complexd>* dev_mhit;
	
	double l2norm = 1./double(n2);
	L2AvgL4ML l2avgl4(l2, l4, sl4, Rmax, l2norm);
	double l4norm = 1./double(n4);
	L4AvgPP<false> l4avgpp(l4, Rmax, l4norm);

	l4->Clear();
	for(int i = 0; i < n4; ++i){
		cout << "Iter of l4: " << i << endl;
		//Update the lattice k4 times freezing spacial links in layers with t multiple of 4
		for(int j = 0; j < k4; ++j){
			mtp4.Run();
			ovr4.Run();
		}
		l2->Clear();
		for(int k = 0; k < n2; ++k){		
			//Update the lattice k2 times freezing spacial links in layers with t multiple of 2
			for(int l = 0; l < k2; ++l){
				mtp2.Run();
				ovr2.Run();	
			}
			//Extract temporal links and apply MultiHit
			dev_mhit = mhitVol.Run();			
			//Calculate tensor T2
			L2ML l2ml(dev_mhit, l2, sl2, Rmax);
			l2ml.Run();
		}
		//Average tensor T2 and Calculate tensor T4
		l2avgl4.Run();	
		
		
		if(PrintResultsAtEveryN4){
			double l4norm1 = 1./double(i+1);
			L4AvgPP<false> l4avgpp1(l4, Rmax, l4norm1);
			Array<complexd>* res = l4avgpp1.Run();
			cout << res << endl;
			delete res;
		}
	}
	delete dev_lat;
	delete dev_mhit;
	delete l2;
	//Average tensor T4 and Calculate P(0)*conj(P(r))	
	Array<complexd>* res = l4avgpp.Run();
	delete l4;

	std::ofstream fileout;
	std::string filename = "Pot_mlevel_" + GetLatticeNameI();
	filename += "_" + ToString(n4) + "_" + ToString(k4);
	filename += "_" + ToString(n2) + "_" + ToString(k2);
	filename += "_" + ToString(metrop) + "_" + ToString(ovrn);
	filename += ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	fileout.precision(12);
	
	for(int r = 0; r < Rmax; ++r){
		cout << r << '\t' << res->at(r) << endl;
		fileout << r << '\t' << res->at(r) << endl;
	}
	
	fileout.close();	
	a0.stop();
	std::cout << "time " << a0.getElapsedTime() << " s" << endl;
	return res;
} 







void MultiLevelField(Array<double> *lat, CudaRNG *rng_state, Array<complexd> **pp, Array<complexd> **ppfield, int n4, int k4, int n2, int k2, int metrop, int ovrn, int Rmax, bool PrintResultsAtEveryN4){
	Timer a0; a0.start();

	cout << "==============================================" << endl;
	cout << "Rmax: " << Rmax << endl;
	cout << "----------------------------------------------" << endl;
	cout << "Level 0:" << endl;
	cout << "\tNº time links per slice: " << 2 << endl;
	cout << "\tNº iterations: " << n2 << endl;
	cout << "\tNº updates: " << k2 << endl;
	cout << "\tNº metropolis updates: " << metrop << endl;
	cout << "\tNº overrelaxation updates: " << ovrn << endl;
	cout << "----------------------------------------------" << endl;
	cout << "Level 1:" << endl;
	cout << "\tNº time links per slice: " << 4 << endl;
	cout << "\tNº iterations: " << n4 << endl;
	cout << "\tNº updates: " << k4 << endl;
	cout << "\tNº metropolis updates: " << metrop << endl;
	cout << "\tNº overrelaxation updates: " << ovrn << endl;
	cout << "==============================================" << endl;
	
	if( Grid(TDir())%4 != 0 ) {
		cout << "Error: Cannot Apply MultiLevel Algorithm...\n Nt is not multiple of 4...\n Exiting..." << endl;
		exit(1);
	}
	Array<double>* dev_lat = new Array<double>(Device);
	dev_lat->Copy(lat);

	int nl2 = Grid(TDir())/2;
	int sl2 = nl2*(Dirs()-1)*Rmax*SpatialVolume();
	Array<complexd> *l2 = new Array<complexd>(Device, sl2);
	int nl4 = Grid(TDir())/4;
	size_t sl4 = nl4*(Dirs()-1)*Rmax*SpatialVolume();
	Array<complexd> *l4 = new Array<complexd>(Device, sl4);
	
	// metropolis and overrelaxation algorithm
	Metropolis_ML<4> mtp4(dev_lat, rng_state, metrop);
	OverRelaxation_ML<4> ovr4(dev_lat, ovrn);
	
	Metropolis_ML<2> mtp2(dev_lat, rng_state, metrop);
	OverRelaxation_ML<2> ovr2(dev_lat, ovrn);
	
	const bool multihit = true;
	Polyakov_Volume<multihit> mhitVol(dev_lat);
	Array<complexd>* dev_mhit;
	
	double l2norm = 1./double(n2);
	L2AvgL4ML l2avgl4(l2, l4, sl4, Rmax, l2norm);
	double l4norm = 1./double(n4);
	L4AvgPP<true> l4avgpp(l4, Rmax, l4norm);

	l4->Clear();
	for(int i = 0; i < n4; ++i){
		cout << "Iter of l4: " << i << endl;
		//Update the lattice k4 times freezing spacial links in layers with t multiple of 4
		for(int j = 0; j < k4; ++j){
			mtp4.Run();
			ovr4.Run();
		}
		l2->Clear();
		for(int k = 0; k < n2; ++k){		
			//Update the lattice k2 times freezing spacial links in layers with t multiple of 2
			for(int l = 0; l < k2; ++l){
				mtp2.Run();
				ovr2.Run();	
			}
			//Extract temporal links and apply MultiHit
			dev_mhit = mhitVol.Run();			
			//Calculate tensor T2
			L2ML l2ml(dev_mhit, l2, sl2, Rmax);
			l2ml.Run();
		}
		//Average tensor T2 and Calculate tensor T4
		l2avgl4.Run();	
		
		
		if(PrintResultsAtEveryN4){
			double l4norm1 = 1./double(i+1);
			L4AvgPP<false> l4avgpp1(l4, Rmax, l4norm1);
			Array<complexd>* res = l4avgpp1.Run();
			cout << res << endl;
			delete res;
		}
	}
	delete dev_lat;
	delete dev_mhit;
	delete l2;
	//Average tensor T4 and Calculate P(0)*conj(P(r))
	*pp = l4avgpp.Run();
	delete l4;
	*ppfield = l4avgpp.getField();

	std::ofstream fileout;
	std::string filename = "Pot_mlevel_" + GetLatticeNameI();
	filename += "_" + ToString(n4) + "_" + ToString(k4);
	filename += "_" + ToString(n2) + "_" + ToString(k2);
	filename += "_" + ToString(metrop) + "_" + ToString(ovrn);
	filename += ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	cout << "Saving data to " << filename << endl;
	fileout << std::scientific;
	fileout.precision(14);
	cout << std::scientific;
	cout << std::setprecision(14);
	
	for(int r = 0; r < Rmax; ++r){
		cout << r << '\t' << (*pp)->at(r) << endl;
		fileout << r << '\t' << (*pp)->at(r) << endl;
	}
	
	fileout.close();
	a0.stop();
	std::cout << "time " << a0.getElapsedTime() << " s" << endl;
}


}
