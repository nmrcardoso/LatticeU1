#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h> 
#include <time.h> 
#include <random>
#include <vector> 
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "timer.h"
#include "cuda_error_check.h"
#include "alloc.h"
#include "reduce_block_1d.h"
#include "complex.h"

#include "parameters.h"
#include "index.h"
#include "staple.h"
#include "random.h"


#include "tune.h"

#include "array.h"
#include "multilevel.h"
#include "lattice_functions.h"

using namespace std;

namespace U1{


#include "multilevel_common.cuh"





template<bool multihit>
__global__ void kernel_l2_multilevel_11(double *lat, complexd *l2, int Rmax, int nl0){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;    
	if(id >= SpatialVolume()) return;	
	
	int parity = 0;
	if(id >= SpatialVolume()/2){
		id -= SpatialVolume()/2;
		parity = 1;
	} 	
	int x[4];
	indexEO(id, parity, x);
	
	int nlayers = Grid(TDir())/nl0;
	for(int r = 0; r < Rmax; ++r){	
		for(int dir = 0; dir < TDir(); dir++){		
			int layer = 0;
			for(int t = 0; t < Grid(TDir()); t+=nl0){
				complexd pl0 = 1.;
				complexd pl1 = 1.;
				for(x[TDir()] = t; x[TDir()] < t+nl0; ++x[TDir()]){
					int newid = indexId(x) >> 1;
					int parity = GetParity(x);
					if(multihit){
						pl0 *= MultiHit(lat, newid, parity, TDir());
					}
					else{
						pl0 *= exp_ir(lat[newid + parity * HalfVolume() + TDir() * Volume()]);
					}
					int xold = x[dir];
					x[dir] = (x[dir] + r) % Grid(dir);
					newid = indexId(x) >> 1;
					parity = GetParity(x);
					if(multihit){
						pl1 *= conj(MultiHit(lat, newid, parity, TDir()));
					}
					else{
						pl1 *= conj(exp_ir(lat[newid + parity * HalfVolume() + TDir() * Volume()]));
					}
									
					x[dir] = xold;
				}			
				complexd pl= pl0 * pl1;			
				int pos = indexIdS(x) + SpatialVolume() * r + SpatialVolume() * Rmax * dir + SpatialVolume() * Rmax * (Dirs()-1) * layer;
				l2[pos] = pl + l2[pos];
				layer++;
			}
		}
	}
}


template<bool multihit>
class L2ML1: Tunable{
private:
	Array<double> *lat;
	Array<complexd> *l2;
	int nl0;
	size_t sl2;
	int Rmax;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_l2_multilevel_11<multihit><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(lat->getPtr(), l2->getPtr(), Rmax, nl0);
}
public:	
   L2ML1(Array<double> *lat, Array<complexd> *l2, size_t sl2, int Rmax, int nl0) : lat(lat), l2(l2), sl2(sl2), Rmax(Rmax), nl0(nl0) {
	size = SpatialVolume();
	timesec = 0.0;  
}
   ~L2ML1(){ };
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_time(){	return timesec;}
   void stat(){	cout << "kernel_l2_multilevel_1:  " <<  get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {
  	l2->Backup();	
  }
  void postTune() {  
	l2->Restore();
 }

};
























__global__ void kernel_l2_multilevel_1(complexd *poly, complexd *l2, int Rmax, int nl0){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;    
	if(id >= SpatialVolume()) return;		
	int x[4];
	indexNOSD(id, x);
	
	int nlayers = Grid(TDir())/nl0;
	for(int r = 0; r < Rmax; ++r){	
		for(int dir = 0; dir < TDir(); dir++){		
			int layer = 0;
			for(int t = 0; t < Grid(TDir()); t+=nl0){
				complexd pl0 = 1.;
				complexd pl1 = 1.;
				for(x[TDir()] = t; x[TDir()] < t+nl0; ++x[TDir()]){
					pl0 *= (poly[indexId(x)]);
					int xold = x[dir];
					x[dir] = (x[dir] + r) % Grid(dir);
					pl1 *= conj(poly[indexId(x)]);
									
					x[dir] = xold;
				}			
				complexd pl= pl0 * pl1;			
				int pos = id + SpatialVolume() * r + SpatialVolume() * Rmax * dir + SpatialVolume() * Rmax * (Dirs()-1) * layer;
				l2[pos] = pl + l2[pos];
				layer++;
			}
		}
	}
}










class L2ML: Tunable{
private:
	Array<complexd> *poly;
	Array<complexd> *l2;
	int nl0;
	size_t sl2;
	int Rmax;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_l2_multilevel_1<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(poly->getPtr(), l2->getPtr(), Rmax, nl0);
}
public:	
   L2ML(Array<complexd> *poly, Array<complexd> *l2, size_t sl2, int Rmax, int nl0) : poly(poly), l2(l2), sl2(sl2), Rmax(Rmax), nl0(nl0) {
	size = SpatialVolume();
	timesec = 0.0;  
}
   ~L2ML(){ };
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_time(){	return timesec;}
   void stat(){	cout << "kernel_l2_multilevel_1:  " <<  get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {
  	l2->Backup();	
  }
  void postTune() {  
	l2->Restore();
 }

};





__global__ void kernel_l2avg_l4_multilevel(complexd *dev_l2, complexd *dev_l4, int Rmax, double l2norm, int nl0, int nl1){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    size_t size = SpatialVolume() * Rmax * (Dirs()-1);
    if(id >= size) return;			
	
	int nl2 = Grid(TDir())/nl0;
	int l1 = nl1/nl0;
	int l4 = 0;
	for(int l2 = 0; l2 < nl2; l2+=l1){
		complexd pl = 1.;
		for(int layer = l2; layer < l2+l1; ++layer){
			int newid = id + size * layer;
			pl *= dev_l2[newid] * l2norm;
		}
		int pos = id + size * l4;
		dev_l4[pos] = pl + dev_l4[pos];
		l4++;	
	}
}


class L2AvgL4ML: Tunable{
private:
	Array<complexd> *l4;
	Array<complexd> *l2;
	int nl0, nl1;
	double l2norm;
	size_t sl4;
	int Rmax;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_l2avg_l4_multilevel<<<tp.grid, tp.block, 0, stream>>>(l2->getPtr(), l4->getPtr(), Rmax, l2norm, nl0, nl1);
}
public:	
   L2AvgL4ML(Array<complexd> *l2, Array<complexd> *l4, size_t sl4, int Rmax, double l2norm, int nl0, int nl1) : l2(l2), l4(l4), sl4(sl4), Rmax(Rmax), l2norm(l2norm), nl0(nl0), nl1(nl1) {
	size = SpatialVolume() * Rmax * (Dirs()-1);
	timesec = 0.0;  
}
   ~L2AvgL4ML(){ };
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_time(){	return timesec;}
   void stat(){	cout << "kernel_l2avg_l4_multilevel:  " <<  get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {
  	l4->Backup();
  }
  void postTune() {  
  	l4->Restore();
 }

};


template<bool savePPspace>
__global__ void kernel_l4avg_Final_multilevel(complexd *dev_l4, complexd *res, complexd *ppSpace, int Rmax, double norm, int nl1){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;    				
	
	int nl4 = Grid(TDir())/nl1;	
	for(int r = 0; r < Rmax; ++r)	{
		complexd pp = 0.;
		if( id < SpatialVolume() ){
			for(int dir = 0; dir < TDir(); dir++){
				complexd pl = 1.;
				for(int l4 = 0; l4 < nl4; ++l4){
					int newid = id + SpatialVolume() * r + SpatialVolume() * Rmax * dir + SpatialVolume() * Rmax * (Dirs()-1) * l4;
					pl *= dev_l4[newid] * norm;
				}
				pp += pl;
				if(savePPspace) ppSpace[id + SpatialVolume() * dir + SpatialVolume() * (Dirs()-1) * r] = pl;
			}
		}
		reduce_block_1d<complexd>(res + r, pp);
		__syncthreads();
	}
}

template<bool savePPspace>
class L4AvgPP: Tunable{
private:
	Array<complexd> *l4;
	Array<complexd> *dev_poly;
	Array<complexd> *poly;
	int nl1;
	int Rmax;
	double norm;
	double l4norm;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return sizeof(complexd); }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	dev_poly->Clear();
	if(savePPspace) kernel_l4avg_Final_multilevel<savePPspace><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(l4->getPtr(), dev_poly->getPtr(), ppSpace->getPtr(), Rmax, l4norm, nl1);
	else kernel_l4avg_Final_multilevel<savePPspace><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(l4->getPtr(), dev_poly->getPtr(), 0, Rmax, l4norm, nl1);
}
public:
	Array<complexd> *ppSpace = 0;
	Array<complexd>* getField(){ return ppSpace; }
	
   L4AvgPP(Array<complexd> *l4, int Rmax, double l4norm, int nl1) : l4(l4), Rmax(Rmax), l4norm(l4norm), nl1(nl1) {
	size = SpatialVolume();
	dev_poly = new Array<complexd>(Device, Rmax);
	if(savePPspace) ppSpace = new Array<complexd>(Device, SpatialVolume() * Rmax * (Dirs()-1));
	poly = new Array<complexd>(Host, Rmax);
	norm = 1. / double(SpatialVolume()*(Dirs()-1));
	timesec = 0.0;  
}
   ~L4AvgPP(){ delete dev_poly; };
   Array<complexd>* Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
	poly->Copy(dev_poly);
	for(int i = 0; i < Rmax; ++i) poly->getPtr()[i] *= norm;
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   Array<complexd>* Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_time(){	return timesec;}
   void stat(){	cout << "kernel_l4avg_Final_multilevel:  " <<  get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};







Array<complexd>* MultiLevel(Array<double> *lat, CudaRNG *rng_state, MLArg *arg, bool PrintResultsAtEveryN4){
	Timer a0; a0.start();
	
	arg->check();	
	arg->print();
	
	
	Array<double>* dev_lat = new Array<double>(Device);
	dev_lat->Copy(lat);

	int nl2 = Grid(TDir())/arg->nLinksLvl0();
	int sl2 = nl2*(Dirs()-1)*arg->Rmax()*SpatialVolume();
	Array<complexd> *l2 = new Array<complexd>(Device, sl2);
	int nl4 = Grid(TDir())/arg->nLinksLvl1();
	size_t sl4 = nl4*(Dirs()-1)*arg->Rmax()*SpatialVolume();
	Array<complexd> *l4 = new Array<complexd>(Device, sl4);
	
	
	// metropolis and overrelaxation algorithm
	Metropolis_ML mtp(dev_lat, rng_state);
	OverRelaxation_ML ovr(dev_lat);
		
	const bool multihit = true;
	Polyakov_Volume<multihit> mhitVol(dev_lat);
	Array<complexd>* dev_mhit;
	
	double l2norm = 1./double(arg->StepsLvl0());
	L2AvgL4ML l2avgl4(l2, l4, sl4, arg->Rmax(), l2norm, arg->nLinksLvl0(), arg->nLinksLvl1());
	double l4norm = 1./double(arg->StepsLvl1());
	L4AvgPP<false> l4avgpp(l4, arg->Rmax(), l4norm, arg->nLinksLvl1());

	l4->Clear();
	for(int i = 0; i < arg->StepsLvl1(); ++i){
		cout << "Iter of l4: " << i << endl;
		//Update the lattice k4 times freezing spacial links in layers with t multiple of 4
		for(int j = 0; j < arg->UpdatesLvl1(); ++j){
			mtp.Run(arg->nUpdatesMetropolis(), arg->nLinksLvl1());
			ovr.Run(arg->nUpdatesOvr(), arg->nLinksLvl1());
		}
		l2->Clear();
		for(int k = 0; k < arg->StepsLvl0(); ++k){		
			//Update the lattice k2 times freezing spacial links in layers with t multiple of 2
			for(int l = 0; l < arg->UpdatesLvl0(); ++l){
				mtp.Run(arg->nUpdatesMetropolis(), arg->nLinksLvl0());
				ovr.Run(arg->nUpdatesOvr(), arg->nLinksLvl0());
			}
			//Extract temporal links and apply MultiHit
			dev_mhit = mhitVol.Run();			
			//Calculate tensor T2
			L2ML l2ml(dev_mhit, l2, sl2, arg->Rmax(), arg->nLinksLvl0());
			//L2ML1<multihit> l2ml(dev_lat, l2, sl2, arg->Rmax(), arg->nLinksLvl0());  // <--- SLOW
			l2ml.Run();
		}
		//Average tensor T2 and Calculate tensor T4
		l2avgl4.Run();	
		
		
		if(PrintResultsAtEveryN4){
			double l4norm1 = 1./double(i+1);
			L4AvgPP<false> l4avgpp1(l4, arg->Rmax(), l4norm1, arg->nLinksLvl1());
			Array<complexd>* res = l4avgpp1.Run();
			cout << res << endl;
			delete res;
		}
	}
	delete dev_lat;
	delete dev_mhit;
	delete l2;
	//Average tensor T4 and Calculate P(0)*conj(P(r))
	Array<complexd> *pp = l4avgpp.Run();
	delete l4;

	std::ofstream fileout;
	std::string filename = "Pot_mlevel_" + GetLatticeNameI();
	filename += "_" + ToString(arg->nLinksLvl1()) + "_" + ToString(arg->StepsLvl1());
	filename += "_" + ToString(arg->UpdatesLvl1()) + "_" + ToString(arg->nLinksLvl0());
	filename += "_" + ToString(arg->StepsLvl0()) + "_" + ToString(arg->UpdatesLvl0());
	filename += "_" + ToString(arg->nUpdatesMetropolis()) + "_" + ToString(arg->nUpdatesOvr());
	filename += "_" + ToString(arg->Rmax());
	filename += ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	cout << "Saving data to " << filename << endl;
	fileout << std::scientific;
	fileout.precision(14);
	cout << std::scientific;
	cout << std::setprecision(14);
	
	for(int r = 0; r < arg->Rmax(); ++r){
		cout << r << '\t' << pp->at(r) << endl;
		fileout << r << '\t' << pp->at(r) << endl;
	}
	
	fileout.close();
	a0.stop();
	std::cout << "time " << a0.getElapsedTime() << " s" << endl;
	
	return pp;
}























std::tuple<Array<complexd>*, Array<complexd>*> MultiLevelField(Array<double> *lat, CudaRNG *rng_state, MLArg *arg, bool PrintResultsAtEveryN4){
	Timer a0; a0.start();
	
	arg->check();	
	arg->print();
	
	
	Array<double>* dev_lat = new Array<double>(Device);
	dev_lat->Copy(lat);

	int nl2 = Grid(TDir())/arg->nLinksLvl0();
	int sl2 = nl2*(Dirs()-1)*arg->Rmax()*SpatialVolume();
	Array<complexd> *l2 = new Array<complexd>(Device, sl2);
	int nl4 = Grid(TDir())/arg->nLinksLvl1();
	size_t sl4 = nl4*(Dirs()-1)*arg->Rmax()*SpatialVolume();
	Array<complexd> *l4 = new Array<complexd>(Device, sl4);
	
	
	// metropolis and overrelaxation algorithm
	Metropolis_ML mtp(dev_lat, rng_state);
	OverRelaxation_ML ovr(dev_lat);
		
	const bool multihit = true;
	Polyakov_Volume<multihit> mhitVol(dev_lat);
	Array<complexd>* dev_mhit;
	
	double l2norm = 1./double(arg->StepsLvl0());
	L2AvgL4ML l2avgl4(l2, l4, sl4, arg->Rmax(), l2norm, arg->nLinksLvl0(), arg->nLinksLvl1());
	double l4norm = 1./double(arg->StepsLvl1());
	L4AvgPP<true> l4avgpp(l4, arg->Rmax(), l4norm, arg->nLinksLvl1());

	l4->Clear();
	for(int i = 0; i < arg->StepsLvl1(); ++i){
		cout << "Iter of l4: " << i << endl;
		//Update the lattice k4 times freezing spacial links in layers with t multiple of 4
		for(int j = 0; j < arg->UpdatesLvl1(); ++j){
			mtp.Run(arg->nUpdatesMetropolis(), arg->nLinksLvl1());
			ovr.Run(arg->nUpdatesOvr(), arg->nLinksLvl1());
		}
		l2->Clear();
		for(int k = 0; k < arg->StepsLvl0(); ++k){		
			//Update the lattice k2 times freezing spacial links in layers with t multiple of 2
			for(int l = 0; l < arg->UpdatesLvl0(); ++l){
				mtp.Run(arg->nUpdatesMetropolis(), arg->nLinksLvl0());
				ovr.Run(arg->nUpdatesOvr(), arg->nLinksLvl0());
			}
			//Extract temporal links and apply MultiHit
			dev_mhit = mhitVol.Run();			
			//Calculate tensor T2
			L2ML l2ml(dev_mhit, l2, sl2, arg->Rmax(), arg->nLinksLvl0());
			//L2ML1<multihit> l2ml(dev_lat, l2, sl2, arg->Rmax(), arg->nLinksLvl0());  // <--- SLOW
			l2ml.Run();
		}
		//Average tensor T2 and Calculate tensor T4
		l2avgl4.Run();	
		
		
		if(PrintResultsAtEveryN4){
			double l4norm1 = 1./double(i+1);
			L4AvgPP<false> l4avgpp1(l4, arg->Rmax(), l4norm1, arg->nLinksLvl1());
			Array<complexd>* res = l4avgpp1.Run();
			cout << res << endl;
			delete res;
		}
	}
	delete dev_lat;
	delete dev_mhit;
	delete l2;
	//Average tensor T4 and Calculate P(0)*conj(P(r))
	Array<complexd> *pp = l4avgpp.Run();
	delete l4;
	Array<complexd> *ppfield = l4avgpp.getField();
	auto data = std::make_tuple(pp, ppfield);

	std::ofstream fileout;
	std::string filename = "Pot_mlevel_" + GetLatticeNameI();
	filename += "_" + ToString(arg->nLinksLvl1()) + "_" + ToString(arg->StepsLvl1());
	filename += "_" + ToString(arg->UpdatesLvl1()) + "_" + ToString(arg->nLinksLvl0());
	filename += "_" + ToString(arg->StepsLvl0()) + "_" + ToString(arg->UpdatesLvl0());
	filename += "_" + ToString(arg->nUpdatesMetropolis()) + "_" + ToString(arg->nUpdatesOvr());
	filename += "_" + ToString(arg->Rmax());
	filename += ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	cout << "Saving data to " << filename << endl;
	fileout << std::scientific;
	fileout.precision(14);
	cout << std::scientific;
	cout << std::setprecision(14);
	
	for(int r = 0; r < arg->Rmax(); ++r){
		cout << r << '\t' << pp->at(r) << endl;
		fileout << r << '\t' << pp->at(r) << endl;
	}
	
	fileout.close();
	a0.stop();
	std::cout << "time " << a0.getElapsedTime() << " s" << endl;
	
	return data;
}




}
