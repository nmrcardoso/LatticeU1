#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h> 
#include <time.h> 
#include <random>
#include <vector> 
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "timer.h"
#include "cuda_error_check.h"
#include "alloc.h"
#include "reduce_block_1d.h"
#include "complex.h"

#include "parameters.h"
#include "index.h"
#include "staple.h"
#include "random.h"


#include "tune.h"

#include "array.h"
#include "multilevel.h"
#include "lattice_functions.h"

using namespace std;

namespace U1{

namespace MLgeneric{


#include "multilevel_generic_common.cuh"





template<bool multihit>
__global__ void kernel_l2_multilevel_11(double *lat, complexd *l2, int radius, int nl0){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;    
	if(id >= SpatialVolume()) return;	
	
	int parity = 0;
	if(id >= SpatialVolume()/2){
		id -= SpatialVolume()/2;
		parity = 1;
	} 	
	int x[4];
	indexEO(id, parity, x);
	
	int nlayers = Grid(TDir())/nl0;
	for(int r = 1; r <= radius; ++r){	
		for(int dir = 0; dir < TDir(); dir++){		
			int layer = 0;
			for(int t = 0; t < Grid(TDir()); t+=nl0){
				complexd pl0 = 1.;
				complexd pl1 = 1.;
				for(x[TDir()] = t; x[TDir()] < t+nl0; ++x[TDir()]){
					int newid = indexId(x) >> 1;
					int parity = GetParity(x);
					if(multihit){
						pl0 *= MultiHit(lat, newid, parity, TDir());
					}
					else{
						pl0 *= exp_ir(lat[newid + parity * HalfVolume() + TDir() * Volume()]);
					}
					int xold = x[dir];
					x[dir] = (x[dir] + r) % Grid(dir);
					newid = indexId(x) >> 1;
					parity = GetParity(x);
					if(multihit){
						pl1 *= conj(MultiHit(lat, newid, parity, TDir()));
					}
					else{
						pl1 *= conj(exp_ir(lat[newid + parity * HalfVolume() + TDir() * Volume()]));
					}
									
					x[dir] = xold;
				}			
				complexd pl= pl0 * pl1;			
				int pos = indexIdS(x) + SpatialVolume() * (r-1) + SpatialVolume() * radius * dir + SpatialVolume() * radius * (Dirs()-1) * layer;
				l2[pos] = pl + l2[pos];
				layer++;
			}
		}
	}
}


template<bool multihit>
class L2ML1: Tunable{
private:
	Array<double> *lat;
	Array<complexd> *l2;
	int nl0;
	size_t sl2;
	int radius;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_l2_multilevel_11<multihit><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(lat->getPtr(), l2->getPtr(), radius, nl0);
}
public:	
   L2ML1(Array<double> *lat, Array<complexd> *l2, size_t sl2, int radius, int nl0) : lat(lat), l2(l2), sl2(sl2), radius(radius), nl0(nl0) {
	size = SpatialVolume();
	timesec = 0.0;  
}
   ~L2ML1(){ };
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_time(){	return timesec;}
   void stat(){	cout << "kernel_l2_multilevel_1:  " <<  get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {
  	l2->Backup();	
  }
  void postTune() {  
	l2->Restore();
 }

};
























__global__ void kernel_l2_multilevel_1(complexd *poly, complexd *l2, int radius, int nl0){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;    
	if(id >= SpatialVolume()) return;		
	int x[4];
	indexNOSD(id, x);
	
	int nlayers = Grid(TDir())/nl0;
	for(int r = 1; r <= radius; ++r){	
		for(int dir = 0; dir < TDir(); dir++){		
			int layer = 0;
			for(int t = 0; t < Grid(TDir()); t+=nl0){
				complexd pl0 = 1.;
				complexd pl1 = 1.;
				for(x[TDir()] = t; x[TDir()] < t+nl0; ++x[TDir()]){
					pl0 *= (poly[indexId(x)]);
					int xold = x[dir];
					x[dir] = (x[dir] + r) % Grid(dir);
					pl1 *= conj(poly[indexId(x)]);
									
					x[dir] = xold;
				}			
				complexd pl= pl0 * pl1;			
				int pos = id + SpatialVolume() * (r-1) + SpatialVolume() * radius * dir + SpatialVolume() * radius * (Dirs()-1) * layer;
				l2[pos] = pl + l2[pos];
				layer++;
			}
		}
	}
}










class L2ML: Tunable{
private:
	Array<complexd> *poly;
	Array<complexd> *l2;
	int nl0;
	size_t sl2;
	int radius;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_l2_multilevel_1<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(poly->getPtr(), l2->getPtr(), radius, nl0);
}
public:	
   L2ML(Array<complexd> *poly, Array<complexd> *l2, size_t sl2, int radius, int nl0) : poly(poly), l2(l2), sl2(sl2), radius(radius), nl0(nl0) {
	size = SpatialVolume();
	timesec = 0.0;  
}
   ~L2ML(){ };
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_time(){	return timesec;}
   void stat(){	cout << "kernel_l2_multilevel_1:  " <<  get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {
  	l2->Backup();	
  }
  void postTune() {  
	l2->Restore();
 }

};





__global__ void kernel_l2avg_l4_multilevel(complexd *dev_l2, complexd *dev_l4, int radius, double l2norm, int nl0, int nl1){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    size_t size = SpatialVolume() * radius * (Dirs()-1);
    if(id >= size) return;			
	
	int nl2 = Grid(TDir())/nl0;
	int l1 = nl1/nl0;
	int l4 = 0;
	for(int l2 = 0; l2 < nl2; l2+=l1){
		complexd pl = 1.;
		for(int layer = l2; layer < l2+l1; ++layer){
			int newid = id + size * layer;
			pl *= dev_l2[newid] * l2norm;
		}
		int pos = id + size * l4;
		dev_l4[pos] = pl + dev_l4[pos];
		l4++;	
	}
}


class L2AvgL4ML: Tunable{
private:
	Array<complexd> *l4;
	Array<complexd> *l2;
	int nl0, nl1;
	double l2norm;
	size_t sl4;
	int radius;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_l2avg_l4_multilevel<<<tp.grid, tp.block, 0, stream>>>(l2->getPtr(), l4->getPtr(), radius, l2norm, nl0, nl1);
}
public:	
   L2AvgL4ML(Array<complexd> *l2, Array<complexd> *l4, size_t sl4, int radius, double l2norm, int nl0, int nl1) : l2(l2), l4(l4), sl4(sl4), radius(radius), l2norm(l2norm), nl0(nl0), nl1(nl1) {
	size = SpatialVolume() * radius * (Dirs()-1);
	timesec = 0.0;  
}
   ~L2AvgL4ML(){ };
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_time(){	return timesec;}
   void stat(){	cout << "kernel_l2avg_l4_multilevel:  " <<  get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() {
  	l4->Backup();
  }
  void postTune() {  
  	l4->Restore();
 }

};


template<bool savePPspace>
__global__ void kernel_l4avg_Final_multilevel(complexd *dev_l4, complexd *res, complexd *ppSpace, int radius, double norm, int nl1){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;    				
	
	int nl4 = Grid(TDir())/nl1;	
	for(int r = 0; r < radius; ++r)	{
		complexd pp = 0.;
		if( id < SpatialVolume() ){
			for(int dir = 0; dir < TDir(); dir++){
				complexd pl = 1.;
				for(int l4 = 0; l4 < nl4; ++l4){
					int newid = id + SpatialVolume() * r + SpatialVolume() * radius * dir + SpatialVolume() * radius * (Dirs()-1) * l4;
					pl *= dev_l4[newid] * norm;
				}
				pp += pl;
				if(savePPspace) ppSpace[id + SpatialVolume() * r + SpatialVolume() * radius * dir] = pl;
			}
		}
		reduce_block_1d<complexd>(res + r, pp);
		__syncthreads();
	}
}

template<bool savePPspace>
class L4AvgPP: Tunable{
private:
	Array<complexd> *l4;
	Array<complexd> *dev_poly;
	Array<complexd> *poly;
	int nl1;
	int radius;
	double norm;
	double l4norm;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return sizeof(complexd); }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	dev_poly->Clear();
	if(savePPspace) kernel_l4avg_Final_multilevel<savePPspace><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(l4->getPtr(), dev_poly->getPtr(), ppSpace->getPtr(), radius, l4norm, nl1);
	else kernel_l4avg_Final_multilevel<savePPspace><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(l4->getPtr(), dev_poly->getPtr(), 0, radius, l4norm, nl1);
}
public:
	Array<complexd> *ppSpace;
	Array<complexd>* getField(){ return ppSpace; }
	
   L4AvgPP(Array<complexd> *l4, int radius, double l4norm, int nl1) : l4(l4), radius(radius), l4norm(l4norm), nl1(nl1) {
	size = SpatialVolume();
	dev_poly = new Array<complexd>(Device, radius);
	if(savePPspace) ppSpace = new Array<complexd>(Device, SpatialVolume() * radius * (Dirs()-1));
	poly = new Array<complexd>(Host, radius);
	norm = 1. / double(SpatialVolume()*(Dirs()-1));
	timesec = 0.0;  
}
   ~L4AvgPP(){ delete dev_poly; };
   Array<complexd>* Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
	poly->Copy(dev_poly);
	for(int i = 0; i < radius; ++i) poly->getPtr()[i] *= norm;
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   Array<complexd>* Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double get_time(){	return timesec;}
   void stat(){	cout << "kernel_l4avg_Final_multilevel:  " <<  get_time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};




Array<complexd>* MultiLevel(Array<double> *lat, CudaRNG *rng_state, int nl0, int nl1, int n4, int k4, int n2, int k2, int metrop, int ovrn, int Rmax, bool PrintResultsAtEveryN4){
	Timer a0; a0.start();
	
	cout << "Rmax: " << Rmax << endl;
	cout << "Level 0:" << endl;
	cout << "\tNº time links per slice: " << nl0 << endl;
	cout << "\tNº iterations: " << n2 << endl;
	cout << "\tNº updates: " << k2 << endl;
	cout << "\tNº metropolis updates: " << metrop << endl;
	cout << "\tNº overrelaxation updates: " << ovrn << endl;
	
	cout << "Level 1:" << endl;
	cout << "\tNº time links per slice: " << nl1 << endl;
	cout << "\tNº iterations: " << n4 << endl;
	cout << "\tNº updates: " << k4 << endl;
	cout << "\tNº metropolis updates: " << metrop << endl;
	cout << "\tNº overrelaxation updates: " << ovrn << endl;
	
	if( Grid(TDir())%nl1 != 0  || Grid(TDir())%nl0 != 0  || nl1%nl0 != 0 ) {
		cout << "Error: Cannot Apply MultiLevel Algorithm...\nExiting..." << endl;
		exit(1);
	}
	Array<double>* dev_lat = new Array<double>(Device);
	dev_lat->Copy(lat);

	int nl2 = Grid(TDir())/nl0;
	int sl2 = nl2*(Dirs()-1)*Rmax*SpatialVolume();
	Array<complexd> *l2 = new Array<complexd>(Device, sl2);
	int nl4 = Grid(TDir())/nl1;
	size_t sl4 = nl4*(Dirs()-1)*Rmax*SpatialVolume();
	Array<complexd> *l4 = new Array<complexd>(Device, sl4);
	
	// metropolis and overrelaxation algorithm
	Metropolis_ML mtp(dev_lat, rng_state);
	OverRelaxation_ML ovr(dev_lat);
		
	const bool multihit = true;
	Polyakov_Volume<multihit> mhitVol(dev_lat);
	Array<complexd>* dev_mhit;
	
	double l2norm = 1./double(n2);
	L2AvgL4ML l2avgl4(l2, l4, sl4, Rmax, l2norm, nl0, nl1);
	double l4norm = 1./double(n4);
	L4AvgPP<false> l4avgpp(l4, Rmax, l4norm, nl1);

	l4->Clear();
	for(int i = 0; i < n4; ++i){
		cout << "Iter of l4: " << i << endl;
		//Update the lattice k4 times freezing spacial links in layers with t multiple of 4
		for(int j = 0; j < k4; ++j){
			mtp.Run(metrop, nl1);
			ovr.Run(ovrn, nl1);
		}
		l2->Clear();
		for(int k = 0; k < n2; ++k){		
			//Update the lattice k2 times freezing spacial links in layers with t multiple of 2
			for(int l = 0; l < k2; ++l){
				mtp.Run(metrop, nl0);
				ovr.Run(ovrn, nl0);
			}
			//Extract temporal links and apply MultiHit
			dev_mhit = mhitVol.Run();			
			//Calculate tensor T2
			L2ML l2ml(dev_mhit, l2, sl2, Rmax, nl0);
			//L2ML1<multihit> l2ml(dev_lat, l2, sl2, Rmax, nl0);  // <--- SLOW
			l2ml.Run();
		}
		//Average tensor T2 and Calculate tensor T4
		l2avgl4.Run();	
		
		
		if(PrintResultsAtEveryN4){
			double l4norm1 = 1./double(i+1);
			L4AvgPP<false> l4avgpp1(l4, Rmax, l4norm1, nl1);
			Array<complexd>* res = l4avgpp1.Run();
			cout << res << endl;
			delete res;
		}
	}
	delete dev_lat;
	delete dev_mhit;
	delete l2;
	//Average tensor T4 and Calculate P(0)*conj(P(r))	
	Array<complexd>* res = l4avgpp.Run();
	delete l4;

	std::ofstream fileout;
	std::string filename = "Pot_mlevel_" + GetLatticeNameI();
	filename += "_" + ToString(n4) + "_" + ToString(k4);
	filename += "_" + ToString(n2) + "_" + ToString(k2);
	filename += "_" + ToString(metrop) + "_" + ToString(ovrn);
	filename += ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	fileout.precision(12);
	
	for(int r = 0; r < Rmax; ++r){
		cout << r+1 << '\t' << res->at(r) << endl;
		fileout << r+1 << '\t' << res->at(r) << endl;
	}
	
	fileout.close();
	a0.stop();
	std::cout << "time " << a0.getElapsedTime() << " s" << endl;	
	return res;
} 







void MultiLevelField(Array<double> *lat, CudaRNG *rng_state, Array<complexd> **pp, Array<complexd> **ppfield, int nl0, int nl1, int n4, int k4, int n2, int k2, int metrop, int ovrn, int Rmax, bool PrintResultsAtEveryN4){
	Timer a0; a0.start();
	
	cout << "Rmax: " << Rmax << endl;
	cout << "Level 0:" << endl;
	cout << "\tNº time links per slice: " << nl0 << endl;
	cout << "\tNº iterations: " << n2 << endl;
	cout << "\tNº updates: " << k2 << endl;
	cout << "\tNº metropolis updates: " << metrop << endl;
	cout << "\tNº overrelaxation updates: " << ovrn << endl;
	
	cout << "Level 1:" << endl;
	cout << "\tNº time links per slice: " << nl1 << endl;
	cout << "\tNº iterations: " << n4 << endl;
	cout << "\tNº updates: " << k4 << endl;
	cout << "\tNº metropolis updates: " << metrop << endl;
	cout << "\tNº overrelaxation updates: " << ovrn << endl;
		
	
	if( Grid(TDir())%nl1 != 0  || Grid(TDir())%nl0 != 0  || nl1%nl0 != 0 ) {
		cout << "Error: Cannot Apply MultiLevel Algorithm...\nExiting..." << endl;
		exit(1);
	}
	Array<double>* dev_lat = new Array<double>(Device);
	dev_lat->Copy(lat);

	int nl2 = Grid(TDir())/nl0;
	int sl2 = nl2*(Dirs()-1)*Rmax*SpatialVolume();
	Array<complexd> *l2 = new Array<complexd>(Device, sl2);
	int nl4 = Grid(TDir())/nl1;
	size_t sl4 = nl4*(Dirs()-1)*Rmax*SpatialVolume();
	Array<complexd> *l4 = new Array<complexd>(Device, sl4);
	
	
	// metropolis and overrelaxation algorithm
	Metropolis_ML mtp(dev_lat, rng_state);
	OverRelaxation_ML ovr(dev_lat);
		
	const bool multihit = true;
	Polyakov_Volume<multihit> mhitVol(dev_lat);
	Array<complexd>* dev_mhit;
	
	double l2norm = 1./double(n2);
	L2AvgL4ML l2avgl4(l2, l4, sl4, Rmax, l2norm, nl0, nl1);
	double l4norm = 1./double(n4);
	L4AvgPP<false> l4avgpp(l4, Rmax, l4norm, nl1);

	l4->Clear();
	for(int i = 0; i < n4; ++i){
		cout << "Iter of l4: " << i << endl;
		//Update the lattice k4 times freezing spacial links in layers with t multiple of 4
		for(int j = 0; j < k4; ++j){
			mtp.Run(metrop, nl1);
			ovr.Run(ovrn, nl1);
		}
		l2->Clear();
		for(int k = 0; k < n2; ++k){		
			//Update the lattice k2 times freezing spacial links in layers with t multiple of 2
			for(int l = 0; l < k2; ++l){
				mtp.Run(metrop, nl0);
				ovr.Run(ovrn, nl0);
			}
			//Extract temporal links and apply MultiHit
			dev_mhit = mhitVol.Run();			
			//Calculate tensor T2
			L2ML l2ml(dev_mhit, l2, sl2, Rmax, nl0);
			//L2ML1<multihit> l2ml(dev_lat, l2, sl2, Rmax, nl0);  // <--- SLOW
			l2ml.Run();
		}
		//Average tensor T2 and Calculate tensor T4
		l2avgl4.Run();	
		
		
		if(PrintResultsAtEveryN4){
			double l4norm1 = 1./double(i+1);
			L4AvgPP<false> l4avgpp1(l4, Rmax, l4norm1, nl1);
			Array<complexd>* res = l4avgpp1.Run();
			cout << res << endl;
			delete res;
		}
	}
	delete dev_lat;
	delete dev_mhit;
	delete l2;
	//Average tensor T4 and Calculate P(0)*conj(P(r))
	*pp = l4avgpp.Run();
	delete l4;
	*ppfield = l4avgpp.getField();

	std::ofstream fileout;
	std::string filename = "Pot_mlevel_" + GetLatticeNameI();
	filename += "_" + ToString(n4) + "_" + ToString(k4);
	filename += "_" + ToString(n2) + "_" + ToString(k2);
	filename += "_" + ToString(metrop) + "_" + ToString(ovrn);
	filename += ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	cout << "Saving data to " << filename << endl;
	fileout << std::scientific;
	fileout.precision(14);
	cout << std::scientific;
	cout << std::setprecision(14);
	
	for(int r = 0; r < Rmax; ++r){
		cout << r+1 << '\t' << (*pp)->at(r) << endl;
		fileout << r+1 << '\t' << (*pp)->at(r) << endl;
	}
	
	fileout.close();
	a0.stop();
	std::cout << "time " << a0.getElapsedTime() << " s" << endl;
}

}


}
