
#include <iostream>
#include <math.h> 
#include <time.h> 
#include <random>
#include <vector> 
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_error_check.h"
#include "enum.h"
#include "parameters.h"


namespace U1{

//static Verbosity verbose = SILENT;
//static Verbosity verbose = DEBUG_VERBOSE;

static TuneMode kerneltune = TUNE_YES;
static Verbosity verbose = VERBOSE;

TuneMode getTuning(){
  return kerneltune;
}
Verbosity getVerbosity(){
  return verbose;
}


void setTuning(TuneMode kerneltunein){
  kerneltune = kerneltunein;
}
void setVerbosity(Verbosity verbosein){
  verbose = verbosein;
}




std::string GetLatticeName(){
	std::string name = "";
	for(int i = 0; i < Dirs(); i++) name += ToString(PARAMS::Grid[i]) + "_";;
	name += "beta_" +  ToString(PARAMS::Beta);
	name += "_xi_" +  ToString(PARAMS::Aniso);	
	name += "_mtr_" +  ToString(PARAMS::metrop);
	name += "_ovr_" +  ToString(PARAMS::ovrn);
	return name;
}



std::string GetLatticeNameI(){
	std::string name = GetLatticeName();
	name += "_iter_" + ToString(PARAMS::iter);
	return name;
}




#define BLOCKSDIVUP(a, b)  (((a)+(b)-1)/(b))


dim3 GetBlockDim(size_t threads, size_t size){
	uint blockx = BLOCKSDIVUP(size, threads);
	dim3 blocks(blockx,1,1);
	return blocks;
}


#define  InlineHostDevice inline  __host__   __device__
#define ConstDeviceMem __constant__

namespace DEVPARAMS{
	ConstDeviceMem   double   Beta;
	ConstDeviceMem   double   Aniso;
	ConstDeviceMem   int DIRS;
	ConstDeviceMem   int TDir;
	ConstDeviceMem   int volume;
	ConstDeviceMem   int half_volume;
	ConstDeviceMem   int spatial_volume;
	ConstDeviceMem   int Grid[4];
}

namespace PARAMS{
	double Beta;
	double Aniso;
	int DIRS;
	int TDir;
	int volume;
	int half_volume;
	int spatial_volume;
	int Grid[4];
	int iter = 0;
	double accept_ratio = 0.;
	int ovrn = 3;
	int metrop = 1;
    hipDeviceProp_t deviceProp;
}

#define memcpyToSymbol(dev, host, type)                                 \
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev),  &host,  sizeof(type), 0, hipMemcpyHostToDevice ));
#define memcpyToArraySymbol(dev, host, type, length)                    \
    cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev),  host,  length * sizeof(type), 0, hipMemcpyHostToDevice ));



void SetupGPU_Parameters(){
	memcpyToSymbol(DEVPARAMS::Beta, PARAMS::Beta, double);
	memcpyToSymbol(DEVPARAMS::volume, PARAMS::volume, int);
	memcpyToSymbol(DEVPARAMS::half_volume, PARAMS::half_volume, int);
	memcpyToSymbol(DEVPARAMS::spatial_volume, PARAMS::spatial_volume, int);
	memcpyToSymbol(DEVPARAMS::DIRS, PARAMS::DIRS, int);
	memcpyToSymbol(DEVPARAMS::TDir, PARAMS::TDir, int);
	memcpyToArraySymbol(DEVPARAMS::Grid, PARAMS::Grid, int, 4); 
	memcpyToSymbol(DEVPARAMS::Aniso, PARAMS::Aniso, double); 
}





}
