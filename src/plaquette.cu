#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h> 
#include <time.h> 
#include <random>
#include <vector> 
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "timer.h"
#include "cuda_error_check.h"
#include "alloc.h"
#include "reduce_block_1d.h"
#include "complex.h"

//#include "special_functions.cuh"

#include "parameters.h"
#include "index.h"
#include "tune.h"
#include "plaquette.h"


using namespace std;


namespace U1{


void plaquette(double *lat, double *plaq){
	for(int i = 0; i < 2; ++i) plaq[i] = 0.;
	for(int parity = 0; parity < 2; ++parity){
		#pragma omp parallel for reduction(+:plaq[:2])
		for(int id = 0; id < HalfVolume(); ++id){
			for(int mu = 0; mu < Dirs() - 1; mu++){	
				double tmp = lat[id + parity * HalfVolume() + mu * Volume()];
				int idmu1 = indexEO_neg(id, parity, mu, 1);
				for (int nu = (mu+1); nu < Dirs(); nu++){			
					double plaqi = tmp;
					plaqi += lat[idmu1 + Volume() * nu];
					plaqi -= lat[indexEO_neg(id, parity, nu, 1) + Volume() * mu];
					plaqi -= lat[id + parity * HalfVolume() + nu * Volume()];
					
					plaq[0] += cos(plaqi);
					plaq[1] += sin(plaqi);	
				}
			}
		}
	}
	int numplaqs = 6; //DIRS=4 3D+1
	if(Dirs()==2) numplaqs = 1.;
	else if(Dirs()==3) numplaqs = 3.;
	double norm = 1. / double(Volume() * numplaqs);
	for(int i = 0; i < 2; ++i) plaq[i] *= norm;
}








__global__ void kernel_plaquette_old(double *lat, complexd *plaq){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;

	complexd plaq0 = 0.;
	
	if( id < HalfVolume() ){
		for(int parity = 0; parity < 2; ++parity){	
			for(int mu = 0; mu < Dirs() - 1; mu++){	
				double tmp = lat[id + parity * HalfVolume() + mu * Volume()];
				int idmu1 = indexEO_neg(id, parity, mu, 1);
				for (int nu = (mu+1); nu < Dirs(); nu++){			
					double plaqi = tmp;
					plaqi += lat[idmu1 + Volume() * nu];
					plaqi -= lat[indexEO_neg(id, parity, nu, 1) + Volume() * mu];
					plaqi -= lat[id + parity * HalfVolume() + nu * Volume()];
					
					plaq0.real() += cos(plaqi);
					plaq0.imag() += sin(plaqi);	
				}
			}
		}
	}
	reduce_block_1d<complexd>(plaq, plaq0);
}



__global__ void kernel_plaquette(double *lat, complexd *plaq){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;

	complexd plaqSS = 0.;
	complexd plaqST = 0.;
	
	if( id < HalfVolume() ){
		for(int parity = 0; parity < 2; ++parity){	
			for(int mu = 0; mu < Dirs() - 1; mu++){	
				double tmp = lat[id + parity * HalfVolume() + mu * Volume()];
				int idmu1 = indexEO_neg(id, parity, mu, 1);
				for (int nu = (mu+1); nu < Dirs(); nu++){			
					double plaqi = tmp;
					plaqi += lat[idmu1 + Volume() * nu];
					plaqi -= lat[indexEO_neg(id, parity, nu, 1) + Volume() * mu];
					plaqi -= lat[id + parity * HalfVolume() + nu * Volume()];
					
					if(mu==TDir() || nu==TDir()) plaqST += 1.0 - exp_ir(plaqi);
					else  plaqSS += 1.0 - exp_ir(plaqi);
				}
			}
		}
	}
	reduce_block_1d<complexd>(plaq, plaqSS);
	reduce_block_1d<complexd>(plaq + 1, plaqST);
}



class Plaquette1: Tunable{
private:
	Array<double>* lat;
	complexd *plaq;
	complexd *dev_plaq;
	double normSS, normST;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return sizeof(complexd); }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	cudaSafeCall(hipMemset(dev_plaq, 0, 2*sizeof(complexd)));
	kernel_plaquette<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(lat->getPtr(), dev_plaq);
}
public:
   Plaquette1(Array<double>* lat, complexd* plaq) : lat(lat), plaq(plaq) {
	size = HalfVolume();
	dev_plaq = (complexd*)dev_malloc(2*sizeof(complexd));
	
	/*int numplaqs = 6; //DIRS=4 3D+1
	if(Dirs()==2) numplaqs = 1.;
	else if(Dirs()==3) numplaqs = 3.;
	norm = 1. / double(Volume() * numplaqs);*/
	
	int npSS = 0;
	int npST = 0;
	for(int mu = 0; mu < Dirs() - 1; mu++)
	for (int nu = (mu+1); nu < Dirs(); nu++){
		if(mu==TDir() || nu==TDir())  npST++;
		else npSS++; 
	}
	//cout << npSS << '\t' << npST << endl;
	normSS = 1. / double(Volume() * npSS);
	normST = 1. / double(Volume() * npST);
	timesec = 0.0;  
}
   ~Plaquette1(){ dev_free(dev_plaq);};
   void Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
	cudaSafeCall(hipMemcpy(plaq, dev_plaq, 2*sizeof(complexd), hipMemcpyDeviceToHost));
	plaq[0] *= normSS;
	//plaq[0] = 1.0 - plaq[0];
	plaq[1] *= normST;
	//plaq[1] = 1.0 - plaq[1];
	//cout << "plaq: " << plaq.real() << '\t' << plaq.imag() << endl;
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
}
   void Run(){ Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double time(){	return timesec;}
   void stat(){	cout << "OverRelaxation:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};








complexd* Plaquette(Array<double> *dev_lat, complexd *plaq, bool print){
	Plaquette1 plaq1(dev_lat, plaq);
	plaq1.Run();
	if(print) cout << "0plaqSS: " << plaq[0] << "\tplaqST: " << plaq[1] << "\tMean: " << (plaq[0] + plaq[1]) * 0.5 << endl;
	if(print) cout << "1plaqSS: " << plaq[0] / Aniso() << "\tplaqST: " << plaq[1] * Aniso() << "\tMean: " << (plaq[0] / Aniso() + plaq[1] * Aniso()) * 0.5 << "\tAction: " << 0.5 * Beta() * (plaq[0] / Aniso() + plaq[1] * Aniso()) << endl;
	return plaq;
} 

}
