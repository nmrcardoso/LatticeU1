#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h> 
#include <time.h> 
#include <random>
#include <vector> 
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "timer.h"
#include "cuda_error_check.h"
#include "alloc.h"
#include "reduce_block_1d.h"
#include "complex.h"


#include "parameters.h"
#include "index.h"


#include "random.h"
#include "staple.h"
#include "update.h"
#include "plaquette.h"

#include "multilevel.h"
#include "tune.h"

using namespace std;


namespace U1{

void polyakov(double *lat, double *poly){
	for(int i = 0; i < 2; ++i) poly[i] = 0.;
	for(int parity = 0; parity < 2; ++parity){
		#pragma omp parallel for reduction(+:poly[:2])
		for(int id = 0; id < SpatialVolume()/2; ++id){
			int x[4];
			indexEO(id, parity, x);
			double tmp = 0.;
			for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()])
				tmp += lat[ indexId(x, TDir()) ];
			poly[0] += cos(tmp);
			poly[1] += sin(tmp);
		}
	}
	double norm = 1. / double(SpatialVolume());
	for(int i = 0; i < 2; ++i) poly[i] *= norm;
}










__global__ void kernel_polyakov(double *lat, complexd *poly){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;

	complexd poly0 = 0.;
	
	if( id < SpatialVolume()/2 ){
		for(int parity = 0; parity < 2; ++parity){		
			int x[4];
			indexEO(id, parity, x);
			double tmp = 0.;
			for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()])
				tmp += lat[ indexId(x, TDir()) ];
			poly0.real() += cos(tmp);
			poly0.imag() += sin(tmp);
		}
	}
	reduce_block_1d<complexd>(poly, poly0);
}



complexd dev_polyakov(double *dev_lat, complexd *dev_poly, int threads, int blocks){
	complexd poly;
	cudaSafeCall(hipMemset(dev_poly, 0, sizeof(complexd)));
	kernel_polyakov<<<blocks, threads, threads*sizeof(complexd)>>>(dev_lat, dev_poly);
	cudaSafeCall(hipMemcpy(&poly, dev_poly, sizeof(complexd), hipMemcpyDeviceToHost));
	poly /= double(SpatialVolume());
	cout << "\t\t" << "L: " << poly.real() << '\t' << poly.imag() << "\t|L|: " << poly.abs() << endl;
	return poly;
} 


using namespace U1;

class Polyakov: Tunable{
private:
	double* lat;
	complexd poly;
	complexd *dev_poly;
	double norm;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return sizeof(complexd); }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	cudaSafeCall(hipMemset(dev_poly, 0, sizeof(complexd)));
	kernel_polyakov<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(lat, dev_poly);
}
public:
   Polyakov(double* lat) : lat(lat) {
	size = SpatialVolume()/2;
	dev_poly = (complexd*)dev_malloc(sizeof(complexd));
	norm = 1. / double(SpatialVolume());
	timesec = 0.0;  
}
   ~Polyakov(){ dev_free(dev_poly);};
   complexd Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
	cudaSafeCall(hipMemcpy(&poly, dev_poly, sizeof(complexd), hipMemcpyDeviceToHost));
	poly *= norm;
    CUDA_SAFE_DEVICE_SYNC();
    CUT_CHECK_ERROR("Kernel execution failed");
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   complexd Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double time(){	return timesec;}
   void stat(){	cout << "OverRelaxation:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};



complexd dev_polyakov(double *dev_lat){
	complexd poly;
	Polyakov pl(dev_lat);
	poly = pl.Run();
	cout << "\t\t" << "L: " << poly.real() << '\t' << poly.imag() << "\t|L|: " << poly.abs() << endl;
	return poly;
} 










__global__ void kernel_polyakov2(double *lat, complexd *poly, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	complexd poly0 = 0.;
	
	if( id < SpatialVolume()/2 ){
		for(int parity = 0; parity < 2; ++parity){		
			int x[4];
			indexEO(id, parity, x);
			
			double tmp = 0.;
			for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()])
				tmp += lat[ indexId(x, TDir()) ];
				
			for(int dir = 0; dir < TDir(); dir++){
				int xx[4];
				indexEO(id, parity, xx);
				xx[dir] = (xx[dir] + radius) % Grid(dir);
			
				double tmp1 = 0.;
				for(xx[TDir()] = 0; xx[TDir()] < Grid(TDir()); ++xx[TDir()])
					tmp1 += lat[ indexId(xx, TDir()) ];
					
					
					
					
				poly0.real() += cos(tmp-tmp1);
				poly0.imag() += sin(tmp-tmp1);
				
			}
		}
	}
	reduce_block_1d<complexd>(poly, poly0);
}



__global__ void kernel_polyakov21(double *lat, complexd *poly, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	complexd poly0 = 0.;
	
	if( id < SpatialVolume()/2 ){
		for(int parity = 0; parity < 2; ++parity){		
			int x[4];
			indexEO(id, parity, x);
			
			complexd tmp = 1.;
			for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()])
				tmp *= exp_ir(lat[ indexId(x, TDir()) ]);
				
			for(int dir = 0; dir < TDir(); dir++){
				int xx[4];
				indexEO(id, parity, xx);
				xx[dir] = (xx[dir] + radius) % Grid(dir);
			
				complexd tmp1 = 1.;
				for(xx[TDir()] = 0; xx[TDir()] < Grid(TDir()); ++xx[TDir()])
					tmp1 *= exp_ir(lat[ indexId(xx, TDir()) ]);
					
					
				poly0 += tmp * conj(tmp1);
					
				
			}
		}
	}
	reduce_block_1d<complexd>(poly, poly0);
}














complexd dev_polyakov2(double *dev_lat, complexd *dev_poly, int radius, int threads, int blocks){
	complexd poly;
	cudaSafeCall(hipMemset(dev_poly, 0, sizeof(complexd)));
	kernel_polyakov2<<<blocks, threads, threads*sizeof(complexd)>>>(dev_lat, dev_poly, radius);
	cudaSafeCall(hipMemcpy(&poly, dev_poly, sizeof(complexd), hipMemcpyDeviceToHost));
	poly /= double(SpatialVolume()*(Dirs()-1));
	//cout << "\t\t" << "L: " << poly.real() << '\t' << poly.imag() << "\t|L|: " << poly.abs() << endl;
	return poly;
} 









__global__ void kernel_polyakov_volume(double *lat, double *poly){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	/*if( id >= SpatialVolume()/2 ) return;
	for(int parity = 0; parity < 2; ++parity){		
		int x[4];
		indexEO(id, parity, x);
		
		double tmp = 0.;
		for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()]){
			//if(id==0&&parity==0) printf("---->%d\n",x[TDir()]);
			tmp += lat[ indexId(x, TDir()) ];
		}
		poly[((x[2] * Grid(1)) + x[1] ) * Grid(0) + x[0]] = tmp;
	}
	*/
	if( id >= SpatialVolume() ) return;
	int parity = 0;
	if( id >= SpatialVolume()/2 ){
		parity = 1;	
		id -= SpatialVolume()/2;
	}	
	int x[4];
	indexEO(id, parity, x);
	
	double tmp = 0.;
	for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()]){
		//if(id==0&&parity==0) printf("---->%d\n",x[TDir()]);
		tmp += lat[ indexId(x, TDir()) ];
	}
	poly[((x[2] * Grid(1)) + x[1] ) * Grid(0) + x[0]] = tmp;

}


__global__ void kernel_poly2(double *poly, complexd *poly2, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	double pl0 = 0.;
	if(id < SpatialVolume()) pl0 = poly[id];			
	int x[3];
	indexNO3D(id, x);
	for(int r = 1; r <= radius; ++r){	
		complexd pl = 0.;
		if(id < SpatialVolume()){
			for(int dir = 0; dir < TDir(); dir++){
				int xold = x[dir];
				x[dir] = (x[dir] + r) % Grid(dir);
				double pl1 = poly[((x[2] * Grid(1)) + x[1] ) * Grid(0) + x[0]];
				pl1 = pl0-pl1;
				pl.real() += cos(pl1);
				pl.imag() += sin(pl1);	
				//pl += exp_ir(pl0) * conj(exp_ir(pl1));				
				x[dir] = xold;
			}
		}				
		reduce_block_1d<complexd>(poly2 + r - 1, pl);
		__syncthreads();
	}
}

__global__ void kernel_poly21(double *poly, complexd *poly2, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	complexd pl0 = 0.;
	if(id < SpatialVolume()) pl0 = exp_ir(poly[id]);			
	int x[3];
	indexNO3D(id, x);
	for(int r = 1; r <= radius; ++r){	
		complexd pl = 0.;
		if(id < SpatialVolume()){
			for(int dir = 0; dir < TDir(); dir++){
				int xold = x[dir];
				x[dir] = (x[dir] + r) % Grid(dir);
				double pl1 = poly[((x[2] * Grid(1)) + x[1] ) * Grid(0) + x[0]];
				//pl1 = pl0-pl1;
				//pl.real() += cos(pl1);
				//pl.imag() += sin(pl1);	
				pl += pl0*conj(exp_ir(pl1));				
				x[dir] = xold;
			}
		}				
		reduce_block_1d<complexd>(poly2 + r - 1, pl);
		__syncthreads();
	}
}


complexd* poly2(double *dev_lat){
	int radius = Grid(0)/2;
	double *dev_poly_vol = (double*)dev_malloc(SpatialVolume()*sizeof(double));
	complexd *dev_poly2 = (complexd*)dev_malloc(radius*sizeof(complexd));
	complexd *poly2 = (complexd*)safe_malloc(radius*sizeof(complexd));
	
	
	int threads = 128;
	//int blocks0 = (HalfVolume() + threads - 1) / threads;
	int blocks0 = (SpatialVolume() + threads - 1) / threads;
	int blocks1 = (SpatialVolume() + threads - 1) / threads;
	size_t smem = threads * sizeof(complexd);
	
	
	kernel_polyakov_volume<<<blocks0, threads>>>(dev_lat, dev_poly_vol);
	//cudaSafeCall(hipMemset(dev_poly2, 0, radius*sizeof(complexd)));
	kernel_poly2<<<blocks1, threads, smem>>>(dev_poly_vol, dev_poly2, radius);
	cudaSafeCall(hipMemcpy(poly2, dev_poly2, radius*sizeof(complexd), hipMemcpyDeviceToHost));
	
	
	

	std::ofstream fileout;
	std::string filename = "Pot_" + GetLatticeName() + ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	fileout.precision(12);
		
	for(int r = 0; r < radius; ++r){
		poly2[r] /= double(SpatialVolume()*(Dirs()-1));
		cout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
		fileout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
	}
	fileout.close();	
	
	dev_free(dev_poly2);
	dev_free(dev_poly_vol);
	//host_free(poly2);
	return poly2;
} 









__global__ void kernel_polyakov_volume_mhit(double *lat, complexd *poly){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
   
	if( id >= SpatialVolume() ) return;
	int parity = 0;
	if( id >= SpatialVolume()/2 ){
		parity = 1;	
		id -= SpatialVolume()/2;
	}	
	int x[4];
	indexEO(id, parity, x);
	
	complexd res = 1.;
	for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()]){
		double W_re, W_im;
		
		int pos = ((((x[3] * Grid(2) + x[2]) * Grid(1)) + x[1] ) * Grid(0) + x[0]) >> 1;
		int oddbit = (x[0] + x[1] + x[2] + x[3]) & 1;

		staple(lat, pos, oddbit, TDir(), W_re, W_im);				
		
		double alpha = sqrt(W_re*W_re+W_im*W_im);
	
		double ba = Beta() * alpha;
		double temp = cyl_bessel_i1(ba)/(cyl_bessel_i0(ba)*alpha);
		//double temp = besseli1(ba)/(besseli0(ba)*alpha);
		complexd val(temp*W_re, -temp*W_im);
		
		res *= val;
	}
	poly[((x[2] * Grid(1)) + x[1] ) * Grid(0) + x[0]] = res;

}

__global__ void kernel_poly2_mhit(complexd *poly, complexd *poly2, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	complexd pl0 = 0.;
	if(id < SpatialVolume()) pl0 = poly[id];			
	int x[3];
	indexNO3D(id, x);
	for(int r = 1; r <= radius; ++r){	
		complexd pl = 0.;
		if(id < SpatialVolume()){
			for(int dir = 0; dir < TDir(); dir++){
				int xold = x[dir];
				x[dir] = (x[dir] + r) % Grid(dir);
				complexd pl1 = poly[((x[2] * Grid(1)) + x[1] ) * Grid(0) + x[0]];
				pl += pl0 * conj(pl1);				
				x[dir] = xold;
			}
		}				
		reduce_block_1d<complexd>(poly2 + r - 1, pl);
		__syncthreads();
	}
}


complexd* poly2_mhit(double *dev_lat){
	int radius = Grid(0)/2;
	complexd *dev_poly_vol = (complexd*)dev_malloc(SpatialVolume()*sizeof(complexd));
	complexd *dev_poly2 = (complexd*)dev_malloc(radius*sizeof(complexd));
	complexd *poly2 = (complexd*)safe_malloc(radius*sizeof(complexd));
		
	int threads = 128;
	//int blocks0 = (HalfVolume() + threads - 1) / threads;
	int blocks0 = (SpatialVolume() + threads - 1) / threads;
	int blocks1 = (SpatialVolume() + threads - 1) / threads;
	size_t smem = threads * sizeof(complexd);
	
	
	kernel_polyakov_volume_mhit<<<blocks0, threads>>>(dev_lat, dev_poly_vol);
	//cudaSafeCall(hipMemset(dev_poly2, 0, radius*sizeof(complexd)));
	kernel_poly2_mhit<<<blocks1, threads, smem>>>(dev_poly_vol, dev_poly2, radius);
	cudaSafeCall(hipMemcpy(poly2, dev_poly2, radius*sizeof(complexd), hipMemcpyDeviceToHost));
	
	std::ofstream fileout;
	std::string filename = "Pot_mhit_" + GetLatticeName() + ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	fileout.precision(12);
		
	for(int r = 0; r < radius; ++r){
		poly2[r] /= double(SpatialVolume()*(Dirs()-1));
		cout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
		fileout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
	}
	
	fileout.close();	
	
	
	
	
	//host_free(poly2);
	dev_free(dev_poly2);
	dev_free(dev_poly_vol);
	return poly2;
} 




















template< bool multihit>
__global__ void kernel_polyakov_volume(double *lat, complexd *poly){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
   
	if( id >= SpatialVolume() ) return;
	int parity = 0;
	if( id >= SpatialVolume()/2 ){
		parity = 1;	
		id -= SpatialVolume()/2;
	}	
	int x[4];
	indexEO(id, parity, x);
	
	complexd res = 1.;
	for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()]){	
		if(multihit){
			double W_re, W_im;				
			int pos = ((((x[3] * Grid(2) + x[2]) * Grid(1)) + x[1] ) * Grid(0) + x[0]) >> 1;
			int oddbit = (x[0] + x[1] + x[2] + x[3]) & 1;
			staple(lat, pos, oddbit, TDir(), W_re, W_im);			
			double alpha = sqrt(W_re*W_re+W_im*W_im);
			double ba = Beta() * alpha;
			double temp = cyl_bessel_i1(ba)/(cyl_bessel_i0(ba)*alpha);
			//double temp = besseli1(ba)/(besseli0(ba)*alpha);
			complexd val(temp*W_re, -temp*W_im);
			res *= val;
		}
		else{
			res *= exp_ir(lat[ indexId(x, TDir()) ]);
		}
	}
	poly[((x[2] * Grid(1)) + x[1] ) * Grid(0) + x[0]] = res;
}

template< bool multihit>
class Polyakov_Vol: Tunable{
private:
	double* lat;
	complexd* poly;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_polyakov_volume<multihit><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(lat, poly);
}
public:
   Polyakov_Vol(double* lat) : lat(lat) {
	size = SpatialVolume();
	poly = (complexd*)dev_malloc(SpatialVolume()*sizeof(complexd));
	timesec = 0.0;  
}
   ~Polyakov_Vol(){ };
   complexd* Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    CUDA_SAFE_DEVICE_SYNC();
    CUT_CHECK_ERROR("Kernel execution failed");
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   complexd* Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double time(){	return timesec;}
   void stat(){	cout << "OverRelaxation:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};





__global__ void kernel_PP(complexd *poly, complexd *res, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	complexd pl0 = 0.;
	if(id < SpatialVolume()) pl0 = poly[id];			
	int x[3];
	indexNO3D(id, x);
	for(int r = 1; r <= radius; ++r){	
		complexd pl = 0.;
		if(id < SpatialVolume()){
			for(int dir = 0; dir < TDir(); dir++){
				int xold = x[dir];
				x[dir] = (x[dir] + r) % Grid(dir);
				complexd pl1 = poly[((x[2] * Grid(1)) + x[1] ) * Grid(0) + x[0]];
				pl += pl0 * conj(pl1);				
				x[dir] = xold;
			}
		}				
		reduce_block_1d<complexd>(res + r - 1, pl);
		__syncthreads();
	}
}




class PP: Tunable{
private:
	complexd* pvol;
	complexd *poly;
	complexd *dev_poly;
	int radius;
	double norm;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return sizeof(complexd); }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	cudaSafeCall(hipMemset(dev_poly, 0, radius*sizeof(complexd)));
	kernel_PP<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(pvol, dev_poly, radius);
}
public:
   PP(complexd *pvol, int radius) : pvol(pvol), radius(radius) {
	size = SpatialVolume();
	dev_poly = (complexd*)dev_malloc(radius*sizeof(complexd));
	poly = (complexd*)safe_malloc(radius*sizeof(complexd));
	norm = 1. / double(SpatialVolume()*(Dirs()-1));
	timesec = 0.0;  
}
   ~PP(){ dev_free(dev_poly);};
   complexd* Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
	cudaSafeCall(hipMemcpy(poly, dev_poly, radius*sizeof(complexd), hipMemcpyDeviceToHost));
	for(int i = 0; i < radius; ++i) poly[i] *= norm;
    CUDA_SAFE_DEVICE_SYNC();
    CUT_CHECK_ERROR("Kernel execution failed");
#ifdef TIMMINGS
	CUDA_SAFE_DEVICE_SYNC( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   complexd* Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double time(){	return timesec;}
   void stat(){	cout << "OverRelaxation:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};




complexd* Poly2(double *lat, bool multihit){
	int radius = Grid(0)/2;
	
	complexd* poly;
	if(multihit){
		Polyakov_Vol<true> pvol(lat);
		poly = pvol.Run();
	}
	else{
		Polyakov_Vol<false> pvol(lat);
		poly = pvol.Run();
	}
	PP pp(poly, radius);
	complexd* poly2 = pp.Run();
	dev_free(poly);
	
	std::ofstream fileout;
	std::string filename = "Pot_" + GetLatticeName() + ".dat";
	if(multihit) filename = "Pot_mhit_" + GetLatticeName() + ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	fileout.precision(12);
		
	for(int r = 0; r < radius; ++r){
		cout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
		fileout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
	}
	
	fileout.close();	
	
	//host_free(poly2);
	return poly2;
} 


}

