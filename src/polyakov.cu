#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h> 
#include <time.h> 
#include <random>
#include <vector> 
#include <fstream>
#include <omp.h>

#include <hip/hip_runtime.h>

#include "timer.h"
#include "cuda_error_check.h"
#include "alloc.h"
#include "reduce_block_1d.h"
#include "complex.h"


#include "parameters.h"
#include "index.h"
#include "array.h"


#include "staple.h"

#include "tune.h"
#include "lattice_functions.h"

using namespace std;


namespace U1{

void polyakov(double *lat, double *poly){
	for(int i = 0; i < 2; ++i) poly[i] = 0.;
	for(int parity = 0; parity < 2; ++parity){
		#pragma omp parallel for reduction(+:poly[:2])
		for(int id = 0; id < SpatialVolume()/2; ++id){
			int x[4];
			indexEO(id, parity, x);
			double tmp = 0.;
			for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()])
				tmp += lat[ indexId(x, TDir()) ];
			poly[0] += cos(tmp);
			poly[1] += sin(tmp);
		}
	}
	double norm = 1. / double(SpatialVolume());
	for(int i = 0; i < 2; ++i) poly[i] *= norm;
}










__global__ void kernel_polyakov(double *lat, complexd *poly){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;

	complexd poly0 = 0.;
	
	if( id < SpatialVolume()/2 ){
		for(int parity = 0; parity < 2; ++parity){		
			int x[4];
			indexEO(id, parity, x);
			double tmp = 0.;
			for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()])
				tmp += lat[ indexId(x, TDir()) ];
			poly0.real() += cos(tmp);
			poly0.imag() += sin(tmp);
		}
	}
	reduce_block_1d<complexd>(poly, poly0);
}



complexd dev_polyakov(double *dev_lat, complexd *dev_poly, int threads, int blocks){
	complexd poly;
	cudaSafeCall(hipMemset(dev_poly, 0, sizeof(complexd)));
	kernel_polyakov<<<blocks, threads, threads*sizeof(complexd)>>>(dev_lat, dev_poly);
	cudaSafeCall(hipMemcpy(&poly, dev_poly, sizeof(complexd), hipMemcpyDeviceToHost));
	poly /= double(SpatialVolume());
	cout << "\t\t" << "L: " << poly.real() << '\t' << poly.imag() << "\t|L|: " << poly.abs() << endl;
	return poly;
} 


using namespace U1;

class CalcPolyakov: Tunable{
private:
	Array<double>* lat;
	complexd poly;
	complexd *dev_poly;
	double norm;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return sizeof(complexd); }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	cudaSafeCall(hipMemset(dev_poly, 0, sizeof(complexd)));
	kernel_polyakov<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(lat->getPtr(), dev_poly);
}
public:
   CalcPolyakov(Array<double>* lat) : lat(lat) {
	size = SpatialVolume()/2;
	dev_poly = (complexd*)dev_malloc(sizeof(complexd));
	norm = 1. / double(SpatialVolume());
	timesec = 0.0;  
}
   ~CalcPolyakov(){ dev_free(dev_poly);};
   complexd Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
	cudaSafeCall(hipMemcpy(&poly, dev_poly, sizeof(complexd), hipMemcpyDeviceToHost));
	poly *= norm;
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   complexd Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double time(){	return timesec;}
   void stat(){	cout << "OverRelaxation:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};



complexd Polyakov(Array<double> *dev_lat, bool print){
	CalcPolyakov pl(dev_lat);
	complexd poly = pl.Run();
	if(print) cout << "L: " << poly.real() << '\t' << poly.imag() << "\t|L|: " << poly.abs() << endl;
	return poly;
} 






__global__ void kernel_polyakov_volume(double *lat, double *poly){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
	if( id >= SpatialVolume() ) return;
	int parity = 0;
	if( id >= SpatialVolume()/2 ){
		parity = 1;	
		id -= SpatialVolume()/2;
	}	
	int x[4];
	indexEO(id, parity, x);
	
	double tmp = 0.;
	for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()]){
		tmp += lat[ indexId(x, TDir()) ];
	}
	poly[indexIdS(x)] = tmp;

}


__global__ void kernel_poly2(double *poly, complexd *poly2, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	double pl0 = 0.;
	if(id < SpatialVolume()) pl0 = poly[id];			
	int x[3];
	indexNOSD(id, x);
	for(int r = 1; r <= radius; ++r){	
		complexd pl = 0.;
		if(id < SpatialVolume()){
			for(int dir = 0; dir < TDir(); dir++){
				int xold = x[dir];
				x[dir] = (x[dir] + r) % Grid(dir);
				double pl1 = poly[indexIdS(x)];
				pl1 = pl0-pl1;
				pl.real() += cos(pl1);
				pl.imag() += sin(pl1);	
				//pl += exp_ir(pl0) * conj(exp_ir(pl1));				
				x[dir] = xold;
			}
		}				
		reduce_block_1d<complexd>(poly2 + r - 1, pl);
		__syncthreads();
	}
}

__global__ void kernel_poly21(double *poly, complexd *poly2, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	complexd pl0 = 0.;
	if(id < SpatialVolume()) pl0 = exp_ir(poly[id]);			
	int x[3];
	indexNOSD(id, x);
	for(int r = 1; r <= radius; ++r){	
		complexd pl = 0.;
		if(id < SpatialVolume()){
			for(int dir = 0; dir < TDir(); dir++){
				int xold = x[dir];
				x[dir] = (x[dir] + r) % Grid(dir);
				double pl1 = poly[indexIdS(x)];
				//pl1 = pl0-pl1;
				//pl.real() += cos(pl1);
				//pl.imag() += sin(pl1);	
				pl += pl0*conj(exp_ir(pl1));				
				x[dir] = xold;
			}
		}				
		reduce_block_1d<complexd>(poly2 + r - 1, pl);
		__syncthreads();
	}
}


complexd* poly2(double *dev_lat){
	int radius = Grid(0)/2;
	double *dev_poly_vol = (double*)dev_malloc(SpatialVolume()*sizeof(double));
	complexd *dev_poly2 = (complexd*)dev_malloc(radius*sizeof(complexd));
	complexd *poly2 = (complexd*)safe_malloc(radius*sizeof(complexd));
	
	
	int threads = 128;
	//int blocks0 = (HalfVolume() + threads - 1) / threads;
	int blocks0 = (SpatialVolume() + threads - 1) / threads;
	int blocks1 = (SpatialVolume() + threads - 1) / threads;
	size_t smem = threads * sizeof(complexd);
	
	
	kernel_polyakov_volume<<<blocks0, threads>>>(dev_lat, dev_poly_vol);
	//cudaSafeCall(hipMemset(dev_poly2, 0, radius*sizeof(complexd)));
	kernel_poly2<<<blocks1, threads, smem>>>(dev_poly_vol, dev_poly2, radius);
	cudaSafeCall(hipMemcpy(poly2, dev_poly2, radius*sizeof(complexd), hipMemcpyDeviceToHost));
	
	
	

	std::ofstream fileout;
	std::string filename = "Pot_" + GetLatticeName() + ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	fileout.precision(12);
		
	for(int r = 0; r < radius; ++r){
		poly2[r] /= double(SpatialVolume()*(Dirs()-1));
		cout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
		fileout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
	}
	fileout.close();	
	
	dev_free(dev_poly2);
	dev_free(dev_poly_vol);
	//host_free(poly2);
	return poly2;
} 









__global__ void kernel_polyakov_volume_mhit(double *lat, complexd *poly){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
   
	if( id >= SpatialVolume() ) return;
	int parity = 0;
	if( id >= SpatialVolume()/2 ){
		parity = 1;	
		id -= SpatialVolume()/2;
	}	
	int x[4];
	indexEO(id, parity, x);
	
	complexd res = 1.;
	for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()]){
		int pos = indexId(x) >> 1;
		int oddbit = GetParity(x);
		res *= MultiHit(lat, pos, oddbit, TDir());
	}
	poly[indexIdS(x)] = res;

}

__global__ void kernel_poly2_mhit(complexd *poly, complexd *poly2, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	complexd pl0 = 0.;
	if(id < SpatialVolume()) pl0 = poly[id];			
	int x[3];
	indexNOSD(id, x);
	for(int r = 1; r <= radius; ++r){	
		complexd pl = 0.;
		if(id < SpatialVolume()){
			for(int dir = 0; dir < TDir(); dir++){
				int xold = x[dir];
				x[dir] = (x[dir] + r) % Grid(dir);
				complexd pl1 = poly[indexIdS(x)];
				pl += pl0 * conj(pl1);				
				x[dir] = xold;
			}
		}				
		reduce_block_1d<complexd>(poly2 + r - 1, pl);
		__syncthreads();
	}
}


complexd* poly2_mhit(double *dev_lat){
	int radius = Grid(0)/2;
	complexd *dev_poly_vol = (complexd*)dev_malloc(SpatialVolume()*sizeof(complexd));
	complexd *dev_poly2 = (complexd*)dev_malloc(radius*sizeof(complexd));
	complexd *poly2 = (complexd*)safe_malloc(radius*sizeof(complexd));
		
	int threads = 128;
	//int blocks0 = (HalfVolume() + threads - 1) / threads;
	int blocks0 = (SpatialVolume() + threads - 1) / threads;
	int blocks1 = (SpatialVolume() + threads - 1) / threads;
	size_t smem = threads * sizeof(complexd);
	
	
	kernel_polyakov_volume_mhit<<<blocks0, threads>>>(dev_lat, dev_poly_vol);
	//cudaSafeCall(hipMemset(dev_poly2, 0, radius*sizeof(complexd)));
	kernel_poly2_mhit<<<blocks1, threads, smem>>>(dev_poly_vol, dev_poly2, radius);
	cudaSafeCall(hipMemcpy(poly2, dev_poly2, radius*sizeof(complexd), hipMemcpyDeviceToHost));
	
	std::ofstream fileout;
	std::string filename = "Pot_mhit_" + GetLatticeName() + ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	fileout.precision(12);
		
	for(int r = 0; r < radius; ++r){
		poly2[r] /= double(SpatialVolume()*(Dirs()-1));
		cout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
		fileout << r+1 << '\t' << poly2[r].real() << '\t' << poly2[r].imag() << endl;
	}
	
	fileout.close();	
	
	
	
	
	//host_free(poly2);
	dev_free(dev_poly2);
	dev_free(dev_poly_vol);
	return poly2;
} 




















template< bool multihit>
__global__ void kernel_polyakov_volume(double *lat, complexd *poly){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
   
	if( id >= SpatialVolume() ) return;
	int parity = 0;
	if( id >= SpatialVolume()/2 ){
		parity = 1;	
		id -= SpatialVolume()/2;
	}	
	int x[4];
	indexEO(id, parity, x);
	
	complexd res = 1.;
	for(x[TDir()] = 0; x[TDir()] < Grid(TDir()); ++x[TDir()]){	
		if(multihit){		
			int pos = indexId(x) >> 1;
			int oddbit = GetParity(x);
			res *= MultiHit(lat, pos, oddbit, TDir());
		}
		else{
			res *= exp_ir(lat[ indexId(x, TDir()) ]);
		}
	}
	poly[indexIdS(x)] = res;
}

template< bool multihit>
class Polyakov_Vol: Tunable{
private:
	Array<double>* lat;
	Array<complexd>* poly;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	kernel_polyakov_volume<multihit><<<tp.grid, tp.block, tp.shared_bytes, stream>>>(lat->getPtr(), poly->getPtr());
}
public:
   Polyakov_Vol(Array<double>* lat) : lat(lat) {
	size = SpatialVolume();
	poly = new Array<complexd>(Device, SpatialVolume() );
	timesec = 0.0;  
}
   ~Polyakov_Vol(){ };
   Array<complexd>* Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   Array<complexd>* Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double time(){	return timesec;}
   void stat(){	cout << "OverRelaxation:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};





__global__ void kernel_PP(complexd *poly, complexd *res, int radius){
    size_t id = threadIdx.x + blockDim.x * blockIdx.x;
    
	complexd pl0 = 0.;
	if(id < SpatialVolume()) pl0 = poly[id];			
	int x[3];
	indexNOSD(id, x);
	for(int r = 1; r <= radius; ++r){	
		complexd pl = 0.;
		if(id < SpatialVolume()){
			for(int dir = 0; dir < TDir(); dir++){
				int xold = x[dir];
				x[dir] = (x[dir] + r) % Grid(dir);
				complexd pl1 = poly[indexIdS(x)];
				pl += pl0 * conj(pl1);				
				x[dir] = xold;
			}
		}				
		reduce_block_1d<complexd>(res + r - 1, pl);
		__syncthreads();
	}
}




class PP: Tunable{
private:
	Array<complexd> *pvol;
	Array<complexd> *poly;
	Array<complexd> *dev_poly;
	int radius;
	double norm;
	int size;
	double timesec;
#ifdef TIMMINGS
    Timer time;
#endif

   unsigned int sharedBytesPerThread() const { return sizeof(complexd); }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
   bool tuneSharedBytes() const { return false; } // Don't tune shared memory
   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   unsigned int minThreads() const { return size; }
   void apply(const hipStream_t &stream){
	TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	dev_poly->Clear();
	kernel_PP<<<tp.grid, tp.block, tp.shared_bytes, stream>>>(pvol->getPtr(), dev_poly->getPtr(), radius);
}
public:
   PP(Array<complexd> *pvol, int radius) : pvol(pvol), radius(radius) {
	size = SpatialVolume();
	dev_poly = new Array<complexd>(Device, radius);
	poly = new Array<complexd>(Host, radius);
	norm = 1. / double(SpatialVolume()*(Dirs()-1));
	timesec = 0.0;  
}
   ~PP(){ delete dev_poly;};
   Array<complexd>* Run(const hipStream_t &stream){
#ifdef TIMMINGS
    time.start();
#endif
	apply(stream);
	poly->Copy(dev_poly);
	for(int i = 0; i < radius; ++i) poly->getPtr()[i] *= norm;
    cudaDevSync();
    cudaCheckError("Kernel execution failed");
#ifdef TIMMINGS
	cudaDevSync( );
    time.stop();
    timesec = time.getElapsedTimeInSec();
#endif
	return poly;
}
   Array<complexd>* Run(){	return Run(0);}
   double flops(){	return ((double)flop() * 1.0e-9) / timesec;}
   double bandwidth(){	return (double)bytes() / (timesec * (double)(1 << 30));}
   long long flop() const { return 0;}
   long long bytes() const{ return 0;}
   double time(){	return timesec;}
   void stat(){	cout << "OverRelaxation:  " <<  time() << " s\t"  << bandwidth() << " GB/s\t" << flops() << " GFlops"  << endl;}
  TuneKey tuneKey() const {
    std::stringstream vol, aux;
    vol << PARAMS::Grid[0] << "x";
    vol << PARAMS::Grid[1] << "x";
    vol << PARAMS::Grid[2] << "x";
    vol << PARAMS::Grid[3];
    aux << "threads=" << size;
    return TuneKey(vol.str().c_str(), typeid(*this).name(), aux.str().c_str());
  }
  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << ")";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }
  void preTune() { }
  void postTune() { }

};




Array<complexd>* Poly2(Array<double> *lat, bool multihit){
	int radius = Grid(0)/2;
	
	Array<complexd>* poly = 0;
	if(multihit){
		Polyakov_Vol<true> pvol(lat);
		poly = pvol.Run();
	}
	else{
		Polyakov_Vol<false> pvol(lat);
		poly = pvol.Run();
	}
	PP pp(poly, radius);
	Array<complexd>* poly2 = pp.Run();
	if(poly) delete poly;
	
	std::ofstream fileout;
	std::string filename = "";
	if(multihit) filename = "Pot_mhit_" + GetLatticeNameI() + ".dat";
	else filename = "Pot_" + GetLatticeNameI() + ".dat";
	fileout.open (filename.c_str());
	if (!fileout.is_open()) {
		std::cout << "Error opening file: " << filename << std::endl;
		exit(1);
	}
	fileout.precision(12);
		
	for(int r = 0; r < radius; ++r){
		cout << r+1 << '\t' << poly2->getPtr()[r].real() << '\t' << poly2->getPtr()[r].imag() << endl;
		fileout << r+1 << '\t' << poly2->getPtr()[r].real() << '\t' << poly2->getPtr()[r].imag() << endl;
	}
	
	fileout.close();	
	
	//host_free(poly2);
	return poly2;
} 


}

